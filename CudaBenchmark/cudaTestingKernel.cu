
#include "hip/hip_runtime.h"

#include <iostream>

// Kind of lame, but just put static file-level variables here for now.
// Pointer to device results array.
float * dev_result = 0;

// Pointer to device data array.
float * dev_data = 0;

// Size of data/result sets (i.e. number of entries in array).
unsigned int testArraySize = 0;

// GPU function to converts the provided dBm value to mW.
// The power in milliwatts (P(mW)) is equal to 1mW times 10 raised by the 
// power in decibel-milliwatts (P(dBm)) divided by 10:
// P(mW) = 1mW * 10 ^ (P(dBm) / 10)
__device__ float convertDbmToMw(const float dBm)
{
  return powf(10.0f, dBm / 10.0f);
}

// Convert dBm to mW kernel to run on the GPU.
__global__ void convertDbmToMwKernal(float * result, const float * data)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  result[i] = convertDbmToMw(data[i]);
}

// Performs cleanup of allocated memory on GPU.
int cleanup()
{
  int status = 0;

  // Free any device allocated memory.
  if (hipFree(dev_result) != hipSuccess)
  {
    status = -1;
    std::cerr << "Failed to free dev_result!\n";
  }

  if (hipFree(dev_data) != hipSuccess)
  {
    status = -1;
    std::cerr << "Failed to free dev_data!\n";
  }

  // Reset the device.
  if (hipDeviceReset() != hipSuccess)
  {
    status = -1;
    std::cerr << "hipDeviceReset failed!\n";
  }

  return status;
}

int setupCudaTestDbmToMw(float * result, float * data, unsigned int size)
{
  int status = 0;

  // Store off test set size.
  testArraySize = size;

  // Choose which GPU to run on, change this on a multi-GPU system.
  hipError_t cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess)
  {
    std::cerr << "hipSetDevice failed!\n";
  }

  // Allocate GPU buffer for output (results).
  cudaStatus = hipMalloc((void**)&dev_result, size * sizeof(int));
  if (cudaStatus != hipSuccess)
  {
    std::cerr << "hipMalloc failed!\n";
  }

  // Allocate GPU buffer for input (data).
  cudaStatus = hipMalloc((void**)&dev_data, size * sizeof(int));
  if (cudaStatus != hipSuccess)
  {
    std::cerr << "hipMalloc failed!\n";
  }

  // Copy input vector from host memory to GPU buffer.
  cudaStatus = hipMemcpy(dev_data, data, size * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess)
  {
    std::cerr << "hipMemcpy failed!\n";
  }

  // If a failure occurred cleanup.
  if (cudaStatus != hipSuccess)
  {
    // Indicate failure occurred.
    status = -1;

    // Attempt to perform cleanup.
    if (cleanup() == -1)
    {
      std::cerr << "cleanup failed!\n";
    }
  }

  return status;
}

// Runs the CUDA test.
int runCudaTest(float * results)
{
  int status = 0;

  // Launch a kernel on the GPU with one thread for each element.

  static const int blockSize = 512;

  int gridSize = (testArraySize + blockSize - 1) / blockSize;

  convertDbmToMwKernal<<<gridSize, blockSize>>>(dev_result, dev_data);

  // Check for any errors launching the kernel
  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess)
  {
    status = -1;
    std::cerr << "convertDbmToMwKernal launch failed " << hipGetErrorString(cudaStatus) << std::endl;
  }

  // cudaDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess)
  {
    status = -1;
    std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus
      << " after launching addKernel!\n";
  }

  // Copy output vector from GPU buffer to host memory.
  cudaStatus = hipMemcpy(results, dev_result, testArraySize * sizeof(float), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess)
  {
    status = -1;
    std::cerr << "hipMemcpy failed!\n";
  }

  // Perform cleanup.
  if (cleanup() == -1)
  {
    status = -1;
    std::cerr << "cleanup failed!\n";
  }

  return status;
}

